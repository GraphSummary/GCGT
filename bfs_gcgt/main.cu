#include <iostream>
#include <fstream>
#include <string>

#include "utils.cuh"
#include "bfs_gcgt.cuh"


int main(int argc,char *argv[]) {

    if (argc != 2) {
		std::cout << "incorrect arguments." << std::endl;
		std::cout << "<input_path>" << std::endl;
        abort();
    }
    std::string input_path(argv[1]);

    hCG hcg;
    hOS hos;

    SIZE_TYPE node_num = load_compressed_graph(input_path, hcg, hos);

    // for lo buf
    hcg.push_back(0);
    hcg.push_back(0);
    hcg.push_back(0);
    hcg.push_back(0);

    printf("%s CGR loaded.", input_path.c_str());

    dCG dcg(hcg);
    dOS dos(hos);
    __dsync__;

    thrust::host_vector<SIZE_TYPE> results(node_num);

    // warm up
    cg_bfs(0, node_num, RAW_PTR(dos), RAW_PTR(dcg), RAW_PTR(results));
    cg_bfs(0, node_num, RAW_PTR(dos), RAW_PTR(dcg), RAW_PTR(results));
    cg_bfs(0, node_num, RAW_PTR(dos), RAW_PTR(dcg), RAW_PTR(results));
    cg_bfs(0, node_num, RAW_PTR(dos), RAW_PTR(dcg), RAW_PTR(results));
    cg_bfs(0, node_num, RAW_PTR(dos), RAW_PTR(dcg), RAW_PTR(results));

    double bfs_time_sum = 0.0;
    int iter_num = 100;
    for (int i = 0; i < iter_num; i++) {
		SIZE_TYPE source = rand() % node_num;

        double cur_time = cg_bfs(source, node_num, RAW_PTR(dos), RAW_PTR(dcg), RAW_PTR(results));
        bfs_time_sum += cur_time;

        SIZE_TYPE unvisited_cnt = thrust::count(results.begin(), results.end(), SIZE_NONE);
		
		printf("[%d]\tsource_node: %d\trunning_time: %.5lf\tvisited_num: %d\n",
			i, source, cur_time, node_num - unvisited_cnt
		);
	}
	
    printf("experiment completed, average running time: %.5lf s.\n", bfs_time_sum / iter_num);

    return 0;
}
